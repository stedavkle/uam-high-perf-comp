#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "practica2\stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "practica2\stb_image_write.h"

#define BLOCK_SIZE 16

__global__ void greyConvert(uint8_t *in, uint8_t *out, int width, int height);

int main(int nargs, char **argv)
{
    int width, height, nchannels;
    struct timeval fin, ini;

    if (nargs < 2)
    {
        printf("Usage: %s <image1> [<image2> ...]\n", argv[0]);
    }
    // For each image
    // Bucle 0
    for (int file_i = 1; file_i < nargs; file_i++)
    {
        printf("[info] Processing %s\n", argv[file_i]);
        /****** Reading file ******/
        uint8_t *rgb_image = stbi_load(argv[file_i], &width, &height, &nchannels, 4);
        if (!rgb_image)
        {
            perror("Image could not be opened");
        }

        /****** Allocating memory ******/
        // - RGB2Grey
        uint8_t *grey_image = (uint8_t *)malloc(width * height);
        if (!grey_image)
        {
            perror("Could not allocate memory");
        }

        // - Filenames 
        for (int i = strlen(argv[file_i]) - 1; i >= 0; i--)
        {
            if (argv[file_i][i] == '.')
            {
                argv[file_i][i] = 0;
                break;
            }
        }

        char *grey_image_filename = 0;
        asprintf(&grey_image_filename, "%s_grey.jpg", argv[file_i]);
        if (!grey_image_filename)
        {
            perror("Could not allocate memory");
            exit(-1);
        }

        /****** Computations ******/
        printf("[info] %s: width=%d, height=%d, nchannels=%d\n", argv[file_i], width, height, nchannels);
        //printf("[info] Checking channels\n");
        if (nchannels != 3 && nchannels != 4)
        {
            printf("[error] Num of channels=%d not supported. Only three (RGB), four (RGBA) are supported.\n", nchannels);
            continue;
        }

        //printf("[info] Starting algorithm\n");

        gettimeofday(&ini,NULL);
        // RGB to grey scale

        int imgSize = sizeof(rgb_image);
        int gImgSize = sizeof(grey_image);

        uint8_t *d_img_in;
        uint8_t *d_img_out;

        hipMalloc((void **) &d_img_in, imgSize);
        hipMalloc((void **) &d_img_out, gImgSize);

        //printf("[cuda] Allocated memory\n");
        //printf("[info] Ready to run Kernel\n");
        hipMemcpy(d_img_in, &rgb_image, imgSize, hipMemcpyHostToDevice);
        //printf("[cuda] Copied image to device\n");

        int blocksPerGrid_x = ceil(width/BLOCK_SIZE);
        int blocksPerGrid_y = ceil(height/BLOCK_SIZE);
        dim3 blocksPerGrid(blocksPerGrid_x, blocksPerGrid_y);
        dim3 threadsPerBlock(16, 16);

        //printf("[cuda] Starting kernel\n");
        greyConvert<<<blocksPerGrid, threadsPerBlock>>>(d_img_in, d_img_out, width, height);
        //printf("[cuda] Kernel finished\n");

        hipMemcpy(&grey_image, d_img_out, gImgSize, hipMemcpyDeviceToHost);

        hipFree(d_img_in), hipFree(d_img_out);

        //printf("[info] Trying to save\n");
        stbi_write_jpg(grey_image_filename, width, height, 1, grey_image, 10);
        free(rgb_image);
        //printf("[info] Done");
        gettimeofday(&fin,NULL);

	    printf("Tiempo: %f\n", ((fin.tv_sec*1000000+fin.tv_usec)-(ini.tv_sec*1000000+ini.tv_usec))*1.0/1000000.0);
        free(grey_image_filename);
    }
}